/**
  This exampis is based on the article titled CUDA Pro Tip: Occupancy API Simplifies Launch Configuration.
  More info on https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/

*/


#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void VectorMultiplicationKernel(int *array, int arrayCount)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < arrayCount)
    {
        array[idx] *= array[idx];
    }
}

void launchMaxOccupancyKernel(int *array, int arrayCount)
{
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, VectorMultiplicationKernel, 0, 0);
    // Round up according to array size
    gridSize = (arrayCount + blockSize - 1) / blockSize;
    printf("Grid size is %d, array count is %d, min grid size is %d\n", gridSize, arrayCount, minGridSize);

    VectorMultiplicationKernel<<< gridSize, blockSize >>>(array, arrayCount);

    hipDeviceSynchronize();

    // calculate theoretical occupancy
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks,
                                                   VectorMultiplicationKernel,
                                                   blockSize, 0);

    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    float occupancy = (float) (maxActiveBlocks * blockSize / props.warpSize) /
            (float)(props.maxThreadsPerMultiProcessor / props.warpSize);

    printf("Device maxThreadsPerMultiProcessor %d\n", props.maxThreadsPerMultiProcessor);
    printf("Device warpSize %d\n", props.warpSize);

    printf("Launched blocks of size %d. Theoretical occupancy: %f\n", blockSize, occupancy);
}

void initializeData(int *array, int count){
    for (int i = 0; i < count; i += 1) {
        array[i] = i;
    }
}

void resetData(int *array, int count){
    for (int i = 0; i < count; i += 1) {
        array[i] = 0;
    }
}

void verifyData(int *array, int count){
    bool isDataCorrect = true;
    // Verify the return data

    for (int i = 0; i < count; i += 1) {
        if (array[i] != i * i) {
            printf("Element %d expected: %d actual %d", i, i *i, array[i]);
            isDataCorrect = false;
        }
    }
    if (isDataCorrect) printf("Data is correct\n");
}

int main()
{
    const int count = 1000000;
    int *array;
    int *dArray;
    int size = count * sizeof(int);

    array = new int[count];

    initializeData(array, count);

    //Maximize occupancy
    hipMalloc(&dArray, size);
    hipMemcpy(dArray, array, size, hipMemcpyHostToDevice);

    resetData(array, count);

    launchMaxOccupancyKernel(dArray, count);

    hipMemcpy(array, dArray, size, hipMemcpyDeviceToHost);
    verifyData(array, count);

    hipFree(dArray);

    delete[] array;

    return 0;
}
